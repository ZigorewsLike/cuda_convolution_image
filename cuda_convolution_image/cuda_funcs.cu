#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <chrono>

#include "cuda_funcs.cuh"

using namespace std;
using namespace std::chrono;

__global__ void add_vector(int* a, int* b, int* c, int N)
{
    long tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    long tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    int l = 0;
    for (int i = 0; i < 1; i++) {
        l += a[tid_x + tid_y * N] + b[tid_x + tid_y * N];
    }
    c[tid_x + tid_y * N] = l;
}

int* calcConvolutionCuda(int N, int M, int* a, int* b) {
    int cuda_count;
    hipGetDeviceCount(&cuda_count);
    printf("Cuda device count = %i\n", cuda_count);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float g_time = 0.0;

    int* c = new int[N * M];

    if (cuda_count == 0) {
        cout << "Cuda device not found!";
        return NULL;
    }

    hipDeviceProp_t info;
    hipGetDeviceProperties(&info, 0);
    int maxCudaTreads = info.maxThreadsPerBlock;
    int maxCudaBlocks = (N * M + maxCudaTreads - 1) / maxCudaTreads;
    printf("threads: %i blocks: %i\n", maxCudaTreads, maxCudaBlocks);

    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc((void**)&dev_a, N * M * sizeof(int));
    hipMalloc((void**)&dev_b, N * M * sizeof(int));
    hipMalloc((void**)&dev_c, N * M * sizeof(int));

    hipMemcpy(dev_a, a, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * M * sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    add_vector << <maxCudaBlocks, maxCudaTreads >> > (dev_a, dev_b, dev_c, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&g_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dev_c, N * M * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf(" time (gpu)= %f mm.\n Calc %i elem\n", g_time, N * M);
    return c;
}

int getCudaDev() {
    int cuda_count;
    hipGetDeviceCount(&cuda_count);
    return cuda_count;
}