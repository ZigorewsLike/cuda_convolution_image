#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <chrono>

#include "cuda_funcs.cuh"

using namespace std;
using namespace std::chrono;

void handleCudaError(hipError_t cudaERR) {
    if (cudaERR != hipSuccess) {
        printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
    }
}

__global__ void add_vector(int* img, int* conv, int* c, int N, int M, int cN)
{
    long tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid_x < N * M * 3) {
            int sum = 0;
            int d = 0; // d Normal
            for (int i = 0; i < cN * cN; i++) {
                int img_x = tid_x / 3 % N;
                int img_y = tid_x / 3 / N;
                int x = tid_x / 3 % N - cN / 2 + i % cN;
                int y = tid_x / 3 / N - cN / 2 + i / cN;
                int thr_x = -5;
                if (x >= 0 && x < N && y >= 0 && y < N) {
                    d += conv[i];
                    thr_x = x + y * N * 3 + tid_x % 3;
                    sum += conv[i] * img[thr_x];
                }
                if(i == 1)
                    printf("channel - %i; imgx = %i, tid_x = %i, imgy = %i\n x = %i, tid_x = %i, new_tid_x = %i, y = %i\n --\n", tid_x % 3, img_x, tid_x, img_y, x, tid_x, thr_x, y);
            }
            sum /= d;
            c[tid_x] = sum;
    }
    
}

int* calcConvolutionCuda(int N, int M, int* img, int* conv, int cN) {
    int cuda_count;
    hipGetDeviceCount(&cuda_count);
    printf("Cuda device count = %i\n", cuda_count);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float g_time = 0.0;

    int* c = new int[N * M];

    if (cuda_count == 0) {
        cout << "Cuda device not found!";
        return NULL;
    }

    hipDeviceProp_t info;
    hipGetDeviceProperties(&info, 0);
    unsigned int maxCudaTreads = info.maxThreadsPerBlock;
    if (maxCudaTreads > N * M*3) {
        maxCudaTreads = N * M*3;
    }
    unsigned int maxCudaBlocks = (N * M*3 + maxCudaTreads - 1) / maxCudaTreads;

    printf("N = %i M = %i\n threads: %i blocks: %i\n", N, M, maxCudaTreads, maxCudaBlocks);

    int* dev_a;
    int* dev_b;
    int* dev_c;

    hipMalloc((void**)&dev_a, 3 * N * M * sizeof(int));
    hipMalloc((void**)&dev_b, N * M * sizeof(int));
    hipMalloc((void**)&dev_c, N * M * sizeof(int));

    handleCudaError(hipMemcpy(dev_a, img, 3 * N * M * sizeof(int), hipMemcpyHostToDevice));
    handleCudaError(hipMemcpy(dev_b, conv, N * M * sizeof(int), hipMemcpyHostToDevice));

    hipEventRecord(start, 0);

    add_vector <<< maxCudaTreads, maxCudaBlocks >>> (dev_a, dev_b, dev_c, N, M, cN);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&g_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dev_c, N * M * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf(" time (gpu)= %f mm.\n Calc %i elem\n", g_time, N * M);
    return c;
}

int getCudaDev() {
    int cuda_count;
    hipGetDeviceCount(&cuda_count);
    return cuda_count;
}