#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <chrono>

#define DELLEXPORT extern "C" __declspec(dllexport)
#pragma comment(linker, "/STACK:2000000")
#pragma comment(linker, "/HEAP:2000000")

using namespace std;
using namespace std::chrono;

void handleCudaError(hipError_t cudaERR) {
    if (cudaERR != hipSuccess) {
        printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
    }
}

__global__ void add_vector(unsigned int* img, int* conv, unsigned int* c, int N, int M, int cN)
{
    long tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid_x < N * M) {
        int sum = 0;
        int d = 0; // d Normal
        for (int i = 0; i < cN * cN; i++) {
            //int img_x = tid_x % N; 
            //int img_y = tid_x / N;
            int x = tid_x % N - cN / 2 + i % cN;
            int y = tid_x / N - cN / 2 + i / cN;
            int thr_x = -5;
            d += conv[i];
            if (x >= -(cN/2) && x <= (N-1)+(cN/2) && y >= -(cN / 2) && y <= (M - 1) + (cN / 2)) {
                if (x <= -1) x = 0;
                else if (x >= N) x = N - 1;
                if (y <= -1) y = 0;
                else if (y >= M) y = M - 1;
                thr_x = x + y * N;
                sum += conv[i] * img[thr_x];
            }
            //if (i == 0)
            //    printf("imgx = %i, tid_x = %i, imgy = %i\n x = %i, tid_x = %i, new_tid_x = %i, y = %i\n Sum = %i img = %i conv = %i\n--\n", img_x, tid_x, img_y, x, tid_x, thr_x, y, sum, img[thr_x], conv[i]);
        }
        if (d != 0) sum /= abs(d);
        if (sum < 0) sum = 0;
        if (sum > 255) sum = 255;
        //printf("tid_x = %i Val = %i\n --\n", tid_x, sum);
        c[tid_x] = sum;
    }
}

DELLEXPORT unsigned int* calcConvolutionCuda(int N, int M, unsigned int* img, int* conv, int cN) {
    int cuda_count;
    hipGetDeviceCount(&cuda_count);
    //printf("\n ---------------------- \nCuda device count = %i\n", cuda_count);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float g_time = 0.0;

    unsigned int* c = new unsigned int[N * M];

    if (cuda_count == 0) {
        //cout << "Cuda device not found!";
        return NULL;
    }
    hipDeviceProp_t info;
    hipGetDeviceProperties(&info, 0);
    unsigned int maxCudaTreads = info.maxThreadsPerBlock;
    if (maxCudaTreads > N * M) {
        maxCudaTreads = N * M;
    }
    unsigned int maxCudaBlocks = (N * M + maxCudaTreads - 1) / maxCudaTreads;

    //printf("Init %i \n", img[0]);

    //printf("N = %i M = %i\n threads: %i blocks: %i\n", N, M, maxCudaTreads, maxCudaBlocks);

    unsigned int* dev_a;
    int* dev_b;
    unsigned int* dev_c;

    //printf("CudaMalloc \n");

    hipMalloc((void**)&dev_a, N * M * sizeof(unsigned int));
    hipMalloc((void**)&dev_b, N * M * sizeof(int));
    hipMalloc((void**)&dev_c, N * M * sizeof(unsigned int));

    //printf("CudaMemcpy \n");

    handleCudaError(hipMemcpy(dev_a, img, N * M * sizeof(unsigned int), hipMemcpyHostToDevice));
    handleCudaError(hipMemcpy(dev_b, conv, N * M * sizeof(int), hipMemcpyHostToDevice));

    hipEventRecord(start, 0);

    //printf("Cuda run \n");

    add_vector << < maxCudaTreads, maxCudaBlocks >> > (dev_a, dev_b, dev_c, N, M, cN);

    //printf("hipMemcpy \n");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&g_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(c, dev_c, N * M * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    //printf("OK \n");

    //printf(" time (gpu)= %f mm.\n Calc %i elem\n", g_time, N * M);
    return c;
}

DELLEXPORT int getCudaDev() {
    int cuda_count;
    hipGetDeviceCount(&cuda_count);
    return cuda_count;
}

